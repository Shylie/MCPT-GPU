#include "hip/hip_runtime.h"
#include "HittableList.h"

#include <hip/hip_runtime.h>
#include <>

__global__ void constructEnvironmentGPU_HittableList(Hittable** this_d, int numHittables, Hittable*** hittables_d)
{
	if (blockIdx.x * blockDim.x + threadIdx.x == 0)
	{
		(*this_d) = new HittableList(numHittables, hittables_d);
	}
}

__global__ void destroyEnvironmentGPU_HittableList(Hittable** this_d)
{
	if (blockIdx.x * blockDim.x + threadIdx.x == 0)
	{
		delete (*this_d);
	}
}

HittableList::HittableList(int numHittables, Hittable** hittables) : numHittables(numHittables), hittables(hittables), hittables_d(new Hittable**[numHittables])
{
#ifndef __CUDA_ARCH__
	for (int i = 0; i < numHittables; i++)
	{
		hittables_d[i] = hittables[i]->GetPtrGPU();
	}

	constructEnvironment();
#endif
}

__device__ HittableList::HittableList(int numHittables, Hittable*** hittables_d) : numHittables(numHittables), hittables_d(hittables_d)
{
}

HittableList::~HittableList()
{
#ifndef __CUDA_ARCH__
	destroyEnvironment();
#endif
}

bool HittableList::Hit(const Ray3& ray, float tMin, float tMax, HitRecord& hRec) const
{
	bool hit = false;
	HitRecord temp;
	for (int i = 0; i < numHittables; i++)
	{
		if ((*(hittables_d[i]))->Hit(ray, tMin, tMax, temp))
		{
			tMax = temp.GetT();
			hRec = temp;
			hit = true;
		}
	}
	return hit;
}

void HittableList::constructEnvironment()
{
	hipMalloc(&this_d, sizeof(Hittable**));
	Hittable*** temp;
	hipMalloc(&temp, numHittables * sizeof(Hittable**));
	device_hittables_d = temp;
	hipMemcpy(temp, hittables_d, numHittables * sizeof(Hittable**), hipMemcpyKind::hipMemcpyHostToDevice);
	constructEnvironmentGPU_HittableList<<<1, 1>>>(this_d, numHittables, temp);
	hipDeviceSynchronize();
}

void HittableList::destroyEnvironment()
{
	hipFree(device_hittables_d);
	destroyEnvironmentGPU_HittableList<<<1, 1>>>(this_d);
	hipFree(this_d);
	hipDeviceSynchronize();
}