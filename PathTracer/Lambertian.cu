#include "hip/hip_runtime.h"
#include "Lambertian.h"

#include <hip/hip_runtime.h>
#include <>

__global__ void constructEnvironmentGPU_Lambertian(Material** this_d, Vec3 albedo)
{
	if (blockIdx.x * blockDim.x + threadIdx.x == 0)
	{
		(*this_d) = new Lambertian(albedo);
	}
}

__global__ void destroyEnvironmentGPU_Lambertian(Material** this_d)
{
	if (blockIdx.x * blockDim.x + threadIdx.x == 0)
	{
		delete (*this_d);
	}
}

Lambertian::Lambertian(Vec3 albedo) : albedo(albedo)
{
#ifndef __CUDA_ARCH__
	constructEnvironment();
#endif
}

Lambertian::~Lambertian()
{
#ifndef __CUDA_ARCH__
	destroyEnvironment();
#endif
}

__host__ __device__ bool Lambertian::Scatter(unsigned int* seed, Ray3& ray, const Vec3& point, const Vec3& normal, Vec3& attenuation) const
{
	Vec3 dir = (normal + Vec3::RandomUnitVector(seed));
	ray = Ray3(point, dir);
	attenuation = albedo;
	return true;
}

__host__ void Lambertian::constructEnvironment()
{
	hipMalloc(&this_d, sizeof(Material**));
	constructEnvironmentGPU_Lambertian<<<1, 1>>>(this_d, albedo);
	hipDeviceSynchronize();
}

__host__ void Lambertian::destroyEnvironment()
{
	destroyEnvironmentGPU_Lambertian<<<1, 1>>>(this_d);
	hipFree(this_d);
	hipDeviceSynchronize();
}