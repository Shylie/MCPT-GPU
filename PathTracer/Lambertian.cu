#include "hip/hip_runtime.h"
#include "Lambertian.h"

#include <hip/hip_runtime.h>
#include <>

__global__ void constructEnvironmentGPU_Lambertian(Material** this_d, Texture** texture_d)
{
	if (blockIdx.x * blockDim.x + threadIdx.x == 0)
	{
		(*this_d) = new Lambertian(texture_d);
	}
}

__global__ void destroyEnvironmentGPU_Lambertian(Material** this_d)
{
	if (blockIdx.x * blockDim.x + threadIdx.x == 0)
	{
		delete (*this_d);
	}
}

Lambertian::Lambertian(Texture* texture) : texture(texture), texture_d(texture->GetPtrGPU())
{
#ifndef __CUDA_ARCH__
	constructEnvironment();
#endif
}

__device__ Lambertian::Lambertian(Texture** texture_d) : texture_d(texture_d)
{
}

Lambertian::~Lambertian()
{
#ifndef __CUDA_ARCH__
	destroyEnvironment();
#endif
}

__host__ __device__ bool Lambertian::Scatter(unsigned int* seed, Ray3& ray, const Vec3& point, const Vec3& normal, Vec3& attenuation) const
{
	Vec3 dir = (normal + Vec3::RandomUnitVector(seed));
	ray = Ray3(point, dir);
#ifdef __CUDA_ARCH__
	attenuation = (*texture_d)->Value(seed, point);
#else
	attenuation = texture->Value(seed, point);
#endif
	return true;
}

__host__ void Lambertian::constructEnvironment()
{
	hipMalloc(&this_d, sizeof(Material**));
	constructEnvironmentGPU_Lambertian<<<1, 1>>>(this_d, texture_d);
	hipDeviceSynchronize();
}

__host__ void Lambertian::destroyEnvironment()
{
	destroyEnvironmentGPU_Lambertian<<<1, 1>>>(this_d);
	hipFree(this_d);
	hipDeviceSynchronize();
}